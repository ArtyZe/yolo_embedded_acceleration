#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <assert.h>
#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void batch_normalize_weights_bias_kernel(float *weights, float * biases, float *rolling_variance, float *rolling_mean, float *scales, 
                                                    float *variance_gpu, float *mean_gpu, int channel_size,int filter_size)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < channel_size){
        if(variance_gpu[i] - rolling_variance[i] > 0.5*rolling_variance[i]){
            // printf("the difference is %f\n", variance_gpu[i] - rolling_variance[i]);
            rolling_variance[i] = variance_gpu[i];
            rolling_mean[i] = mean_gpu[i];
        }
        biases[i] = biases[i] - scales[i] * rolling_mean[i] / (sqrtf(rolling_variance[i]) + .000001f);

        int j;
        for (j = 0; j < filter_size; ++j) {
            int w_index = i*filter_size + j;
            weights[w_index] = weights[w_index] * scales[i] / (sqrtf(rolling_variance[i]) + .000001f);
        }
    }
}

void batch_normalize_weights_bias_gpu(float *weights_gpu, float * bias_gpu, float *rolling_variance_gpu, float *rolling_mean_gpu, float *scale_gpu, 
                                      float *variance_gpu, float *mean_gpu, int channel_size,int filter_size){
    int N = channel_size;
    batch_normalize_weights_bias_kernel<<<cuda_gridsize(N), BLOCK>>>(weights_gpu, bias_gpu, rolling_variance_gpu, rolling_mean_gpu, scale_gpu,
                                                                     variance_gpu, mean_gpu, channel_size, filter_size);
}

__global__ void prune_kernel(int N, float *weights,float *update_weights, float threshold, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        if (fabs(weights[i*INCX])<threshold){
            weights[i*INCX]=0;
            update_weights[i*INCX] = 0;
        }
    }
}

void prune_gpu(int N, float * X, float * Y, float threhold,int INCY)
{
    prune_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X,  Y,threhold, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void set_zero_kernel(float *input, int size)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < size){
        input[i] = 0;
    }
}

void set_zero_gpu(float * X, int N)
{
    set_zero_kernel<<<cuda_gridsize(N), BLOCK>>>(X,  N);
    check_error(hipPeekAtLastError());
}

__global__ void backward_batch_normalize_weights_kernel(int N, float *weights_updates, float *variance, float *scales, int filters, int spatial)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
        int index = i/spatial%filters;
        weights_updates[i] = weights_updates[i]*scales[index]/(sqrt(variance[index]) + .000001f);
            // weights_updates[weights_index] = weights_updates[weights_index]/(sqrt(variance[i]) + .000001f);
	}
}

void backward_batch_normalize_weights_gpu(float *weights_updates, float *variance, float *scales, int filters, int spatial)
{
    int N = filters*spatial;
    backward_batch_normalize_weights_kernel<<<cuda_gridsize(N), BLOCK>>>(N, weights_updates, variance, scales, filters, spatial);
}

__global__ void backward_scale_quant_kernel(int N, float *x_norm, float *weights_update, float *bias_update,  float *mean, float *variance, 
                                            float *rolling_variance, int batch, int spatial, float *scale, float *scale_updates)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index < N*batch){
        for(int j = 0; j < spatial; ++j){
			int weights_index = index*spatial + j;
            scale_updates[index] += weights_update[weights_index]*x_norm[weights_index]/(sqrt(rolling_variance[index]) + .000001f);              
        }
        scale_updates[index] = scale_updates[index] - bias_update[index] *mean[index]/(sqrt(variance[index]) + .000001f);
    }
}

void backward_scale_quant_gpu(float *x_norm, float *weights_update, float *bias_update, float *mean, float *variance, float *rolling_variance,
                              int batch, int channel, int spatial, float *scale, float *scale_updates)
{
    int N = channel;
    backward_scale_quant_kernel<<<N, BLOCK>>>(N, x_norm, weights_update, bias_update, mean, variance, rolling_variance, batch, spatial, scale, scale_updates);
    check_error(hipPeekAtLastError());
}

__global__ void backward_batch_normalize_bias_kernel(float *bias_updates, float *rolling_variance, float *scales, int filters)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < filters){
        bias_updates[i] = (scales[i]*bias_updates[i])/(sqrt(rolling_variance[i]) + .000001f);
        // bias_updates[i] = (bias_updates[i])/(sqrt(rolling_variance[i]) + .000001f);
	}
}

void backward_batch_normalize_bias_gpu(float *bias_updates, float *rolling_variance, float *scales, int filters)
{
    int N = filters;
    backward_batch_normalize_bias_kernel<<<cuda_gridsize(N), BLOCK>>>(bias_updates, rolling_variance, scales, filters);
}



/*************************************************************************************************************************
                        This funtion is main to realize the fake quantization in the paper of

                                "Quantization and Training of Neural Networks for Efficient 
                                        Integer-Arithmetic-Only Inference"
                        
                         We propose an approach that simulates quantization effects in the 
                         forward pass of training. Backpropagation still happens as usual, 
                             and all weights and biases are stored in floating point
 *************************************************************************************************************************/
 __global__ void FakeQuantWithMinMaxChannel_kernel(int size_channel, float *inputs, uint8_t *input_int8, int size_feature, float *min_activ_value, float *max_activ_value, 
                                                   float *quantzation_scale, uint8_t *quantization_zero_point, int func_type, float decay) 
{
    //printf("run in gpu \n");
    //printf("channels %d\n", size_channel);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < size_channel) {
        printf("================================\n");
        //Calculate min and max value of each kernel
        float min_thisKernel_value = inputs[0];
        float max_thisKernel_value = inputs[0];
        int quant_min = -128; 
        int quant_max = 127;
        printf("------------------------------\n");
        for(int j = 0; j < size_feature; ++j){
            int index = i*size_feature+j;
            // printf("index = %d, input = %f\n", index, inputs[index]);
            max_thisKernel_value = max(inputs[index], max_thisKernel_value);
            min_thisKernel_value = min(inputs[index], min_thisKernel_value);
        }
        printf("max value is %f, min value is %f\n",*max_activ_value, *min_activ_value);

        //If this layer is activation, you need to update the min and max value with EMA 
        if(func_type == ACTIV_QUANT){
            *min_activ_value = *min_activ_value - ((*min_activ_value - min_thisKernel_value) * (1- decay));
            *max_activ_value = *max_activ_value - ((*max_activ_value - max_thisKernel_value) * (1- decay));
            // max_thisKernel_value = *max_activ_value;
            // min_thisKernel_value = *min_activ_value;
        }
        // printf("max value is %f, min value is %f\n",max_thisKernel_value, min_thisKernel_value);
        // If min and max are both zero, we should just return zero.
        if (min_thisKernel_value == 0.0f && max_thisKernel_value == 0.0f) {
            printf("ERROR, maybe you give an empty input array\n");
        }

        float nudged_scale = 0.0f;
        //this is really nudge function
        const float quant_min_float = (float)quant_min;
        const float quant_max_float = (float)quant_max;
        nudged_scale = (max_thisKernel_value - min_thisKernel_value) / (quant_max_float - quant_min_float);
        const double initial_zero_point = quant_min - min_thisKernel_value / nudged_scale;
        //printf("the scale is %f\n", nudged_scale);
        //printf("get the zero point start\n");
        //Store the S3 for activ quantization, convenient for us to quantization input in inference process
        quantzation_scale[i] = nudged_scale;
        uint8_t nudged_zero_point = 0;
        if (initial_zero_point < quant_min) {
            nudged_zero_point = quant_min;
        } else if (initial_zero_point > quant_max) {
            nudged_zero_point = quant_max;
        } else {
            nudged_zero_point = round(initial_zero_point);
        }

        quantization_zero_point[i] = nudged_zero_point;
        //printf("get the zero point success\n");
        const float nudged_scale_repl = nudged_scale;
        for(int k = 0; k < size_feature; ++k){
            int index_kernel = i*size_feature+k;
            //float temp = inputs[index_kernel];
            float clamped = max(min_thisKernel_value, min(max_thisKernel_value, inputs[index_kernel]));
            float clamped_shifted = clamped - min_thisKernel_value;
            if(func_type == WEIGHT_QUANT){
                input_int8[index_kernel] = round(clamped_shifted / nudged_scale_repl + 0.01f);
            }
            inputs[index_kernel] = round(clamped_shifted / nudged_scale_repl + 0.01f) * nudged_scale_repl + min_thisKernel_value;
            //printf("the diff is %f\n", inputs[index_kernel] - temp);
        }
    }
}

void FakeQuantWithMinMaxChannel_gpu(int channel_size, float *inputs, uint8_t *input_int8, int size, float *min_value, float *max_value, 
                                    float *quant_scale, uint8_t *quant_zero_point, int layer_type, float decay)
{
    int N =channel_size;
    FakeQuantWithMinMaxChannel_kernel<<<cuda_gridsize(N), BLOCK>>>(N, inputs, input_int8, size, min_value, max_value, quant_scale, quant_zero_point, layer_type, decay);
    check_error(hipPeekAtLastError());
}

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}

void scale_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index]*x_norm[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) scale_updates[filter] += part[i];
    }
}

void backward_scale_gpu(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    backward_scale_kernel<<<n, BLOCK>>>(x_norm, delta, batch, n, size, scale_updates);
    check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

void add_bias_gpu(float *output, float *biases, int batch, int n, int size)
{
    int num = n*size*batch;

    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n) return;
    int b;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        int i = b*n + index;
        sum += delta[i];
    }
    bias_updates[index] += sum;
}

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
    }
}

void backward_bias_gpu(float *bias_updates, float *delta, int batch, int n, int size)
{
    if(size == 1){
        backward_bias_conn_kernel<<<cuda_gridsize(n), BLOCK>>>(bias_updates, delta, batch, n);
    }else{
        backward_bias_kernel<<<n, BLOCK>>>(bias_updates, delta, batch, n, size);
    }
    check_error(hipPeekAtLastError());
}

/*
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;
    
    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    int b, i;
    for(b = 0; b <  batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
    }
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    for(b = 0; b <  batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += - scale * sum * output[i2] / norm;
            delta[i2] += - scale * sum * output[i1] / norm;
        }
    }
}

void dot_error_gpu(layer l)
{
    dot_kernel<<<cuda_gridsize(l.n*l.n), BLOCK>>>(l.output_gpu, l.dot, l.batch, l.n, l.out_w * l.out_h, l.delta_gpu);
    check_error(hipPeekAtLastError());
}
*/

#ifdef MASK
__global__ void mask_weights_kernel(int N, int channel, int size, float *weights, float *weights_mask, float *weights_mask_binary, float threhold, int key)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
		int s;
		
		int c = i%channel;  //channel
		i /= channel;
		int b = i;  //batch
		int mask_index = b*channel + c;  
		for(s = 0; s < size*size; s++){
			int weight_index = (b*channel + c)*size*size + s;
#if 0			
            int zero_num = 0;            
            if(weights_mask[mask_index] != 1){
					printf("the mask is %f, zero num is %d\n", weights_mask[mask_index], zero_num);
					zero_num = zero_num + 1;
			}
#endif			
			weights[weight_index] *= weights_mask[mask_index];
		}
	}
}

void mask_weights_gpu(int N, int channel, int size, float * X, float * Y, float * Z, float threhold, int key)
{

    mask_weights_kernel<<<cuda_gridsize(N), BLOCK>>>(N, channel, size, X, Y, Z, threhold, key);
    check_error(hipPeekAtLastError());
}

__global__ void mask_backward_kernel(int N, int channel, int size, float *weights_result, float *weight_updates, float *weights_mask, float *mask_updates)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
		int s;
		float sum =0;
		int c = i%channel;	//channel
		i /= channel;
		int b = i;	//batch
		int mask_index = b*channel + c;
		for(s = 0; s < size*size; s++){
            int weight_index = (b*channel + c)*size*size + s;
            sum += weight_updates[weight_index]*weights_result[weight_index];
		}
		mask_updates[mask_index] += sum;
		if(mask_updates[mask_index]<0){
			mask_updates[mask_index]= -1 * mask_updates[mask_index];
		}
    }
}


void mask_backward_gpu(int N, int channel, int size, float * X, float * Y, float * Z, float *updates)
{
    mask_backward_kernel<<<cuda_gridsize(N), BLOCK>>>(N, channel, size, X, Y, Z, updates);
    check_error(hipPeekAtLastError());
}

__global__ void mask_update_kernel(int N, int channel, int size, float *weights_mask, float *mask_updates, float threshold)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
		int c = i%channel;	//channel
		i /= channel;
		int b = i;	//batch
		int mask_index = b*channel + c;
		if(mask_updates[mask_index] >= threshold){
			weights_mask[mask_index] = 1;
		}else{
			weights_mask[mask_index] = 0;
		}
    }
}

void mask_update_gpu(int N, int channel, int size, float * X, float * Y, float threshold)
{
    mask_update_kernel<<<cuda_gridsize(N), BLOCK>>>(N, channel, size, X, Y, threshold);
    check_error(hipPeekAtLastError());
}

#endif

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    float mhat = m[index] / (1.f - powf(B1, t));
    float vhat = v[index] / (1.f - powf(B2, t));
    
    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}

void adam_gpu(int n, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    adam_kernel<<<cuda_gridsize(n), BLOCK>>>(n, x, m, v, B1, B2, rate, eps, t);
    check_error(hipPeekAtLastError());
}

void adam_update_gpu(float *w, float *d, float *m, float *v, float B1, float B2, float eps, float decay, float rate, int n, int batch, int t)
{
    scal_gpu(n, B1, m, 1);
    scal_gpu(n, B2, v, 1);
    axpy_gpu(n, -decay*batch, w, 1, d, 1);

    axpy_gpu(n, (1-B1), d, 1, m, 1);
    mul_gpu(n, d, 1, d, 1);
    axpy_gpu(n, (1-B2), d, 1, v, 1);

    adam_gpu(n, w, m, v, B1, B2, rate, eps, t);
    fill_gpu(n, 0, d, 1);
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    x[index] = (x[index] - mean[f])/(sqrtf(variance[f] + .00001f));
}

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    delta[index] = delta[index] * 1.f/(sqrtf(variance[f] + .00001f)) + variance_delta[f] * 2.f * (x[index] - mean[f]) / (spatial * batch) + mean_delta[f]/(spatial*batch);
}

void normalize_delta_gpu(float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    size_t N = batch*filters*spatial;
    normalize_delta_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, mean_delta, variance_delta, batch, filters, spatial, delta);
    check_error(hipPeekAtLastError());
}

__global__ void  variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    variance_delta[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance_delta[i] += delta[index]*(x[index] - mean[i]);
        }
    }
    variance_delta[i] *= -.5f * powf(variance[i] + .00001f, (float)(-3.f/2.f));
}

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    sum[i] = 0;
    for(k = 0; k < n; ++k){
        sum[i] += x[k*groups + i];
    }
}

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? delta[index] : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        mean_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.f/sqrtf(variance[filter] + .00001f));
    }
}

__global__ void  fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? delta[index]*(x[index] - mean[filter]) : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        variance_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -.5f * powf(variance[filter] + .00001f, (float)(-3.f/2.f));
    }
}


__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean_delta[i] = 0;
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            mean_delta[i] += delta[index];
        }
    }
    mean_delta[i] *= (-1.f/sqrtf(variance[i] + .00001f));
}

void mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    mean_delta_kernel<<<cuda_gridsize(filters), BLOCK>>>(delta, variance, batch, filters, spatial, mean_delta);
    check_error(hipPeekAtLastError());
}

void fast_mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    fast_mean_delta_kernel<<<filters, BLOCK>>>(delta, variance, batch, filters, spatial, mean_delta);
    check_error(hipPeekAtLastError());
}

void fast_variance_delta_gpu(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    fast_variance_delta_kernel<<<filters, BLOCK>>>(x, delta, mean, variance, batch, filters, spatial, variance_delta);
    check_error(hipPeekAtLastError());
}

__global__ void  mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f/(batch * spatial);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            mean[i] += x[index];
        }
    }
    mean[i] *= scale;
}

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f/(batch * spatial - 1);
    int j,k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    variance[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance[i] += powf((x[index] - mean[i]), 2);
        }
    }
    variance[i] *= scale;
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_index = i;
    int in_w = i%w;
    i = i/w;
    int in_h = i%h;
    i = i/h;
    int in_c = i%c;
    i = i/c;
    int b = i%batch;

    int out_c = c/(stride*stride);

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w*stride + offset % stride;
    int h2 = in_h*stride + offset / stride;
    //printf("%d\n", offset);
    int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

   // printf("%d %d %d\n", w2, h2, c2);
    //printf("%d %d\n", in_index, out_index);
    //if(out_index >= N || out_index < 0) printf("bad bad bad \n");

    if(forward) out[out_index] = x[in_index];
    else out[in_index] = x[out_index];
    //if(forward) out[1] = x[1];
    //else out[0] = x[0];
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX];
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX]));
}

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        if((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) X[i*INCX] = 0;
    }
}

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] += ALPHA;
}

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] *= ALPHA;
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void fill_kernel_uint8(int N, uint8_t ALPHA, uint8_t *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] *= X[i*INCX];
}


void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int b = index / spatial;
    int i = index % spatial;
    int f;
    float sum = 0;
    for(f = 0; f < filters; ++f){
        int index = b*filters*spatial + f*spatial + i;
        sum += powf(x[index], 2);
    }
    sum = sqrtf(sum);
    if(sum == 0) sum = 1;
    //printf("%f\n", sum);
    for(f = 0; f < filters; ++f){
        int index = b*filters*spatial + f*spatial + i;
        x[index] /= sum;
        dx[index] = (1 - x[index]) / sum;
    }
}

void l2normalize_gpu(float *x, float *dx, int batch, int filters, int spatial)
{
    size_t N = batch*spatial;
    l2norm_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, dx, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}

__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? x[index] : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        mean[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean[filter] += local[i];
        }
        mean[filter] /= spatial * batch;
    }
}

__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? powf((x[index] - mean[filter]), 2) : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        variance[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
    }
}

void fast_mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    fast_mean_kernel<<<filters, BLOCK>>>(x, batch, filters, spatial, mean);
    check_error(hipPeekAtLastError());
}

void fast_variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    fast_variance_kernel<<<filters, BLOCK>>>(x, mean, batch, filters, spatial, variance);
    check_error(hipPeekAtLastError());
}


void mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    mean_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, batch, filters, spatial, mean);
    check_error(hipPeekAtLastError());
}

void variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    variance_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, mean, batch, filters, spatial, variance);
    check_error(hipPeekAtLastError());
}

void axpy_gpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    axpy_gpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

void pow_gpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

void axpy_gpu_offset(int N, float ALPHA, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

void copy_gpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_gpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

void mul_gpu(int N, float * X, int INCX, float * Y, int INCY)
{
    mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

void copy_gpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_s = i%spatial;
    i = i/spatial;
    int in_c = i%layers;
    i = i/layers;
    int b = i;

    int i1 = b*layers*spatial + in_c*spatial + in_s;
    int i2 = b*layers*spatial + in_s*layers +  in_c;

    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}

void flatten_gpu(float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int size = spatial*batch*layers;
    flatten_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, spatial, layers, batch, forward, out);
    check_error(hipPeekAtLastError());
}

void reorg_gpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int size = w*h*c*batch;
    reorg_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, w, h, c, batch, stride, forward, out);
    check_error(hipPeekAtLastError());
}

__global__ void mask_kernel(int n,  float *x, float mask_num, float *mask, float val)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n && mask[i] == mask_num) x[i] = val;
}

void mask_gpu(int N, float * X, float mask_num, float * mask, float val)
{
    mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask, val);
    check_error(hipPeekAtLastError());
}

__global__ void scale_mask_kernel(int n,  float *x, float mask_num, float *mask, float scale)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n && mask[i] == mask_num) x[i] *= scale;
}

void scale_mask_gpu(int N, float * X, float mask_num, float * mask, float scale)
{
    scale_mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask, scale);
    check_error(hipPeekAtLastError());
}

void const_gpu(int N, float ALPHA, float * X, int INCX)
{
    const_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

void constrain_gpu(int N, float ALPHA, float * X, int INCX)
{
    constrain_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}


void add_gpu(int N, float ALPHA, float * X, int INCX)
{
    add_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

void scal_gpu(int N, float ALPHA, float * X, int INCX)
{
    scal_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

void supp_gpu(int N, float ALPHA, float * X, int INCX)
{
    supp_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

void fill_gpu(int N, float ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

void fill_gpu_uint8(int N, uint8_t ALPHA, uint8_t * X, int INCX)
{
    fill_kernel_uint8<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] = s1*out[out_index] + s2*add[add_index];
    //out[out_index] += add[add_index];
}

void shortcut_gpu(int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1/w2;
    int sample = w2/w1;
    assert(stride == h1/h2);
    assert(sample == h2/h1);
    if(stride < 1) stride = 1;
    if(sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    shortcut_kernel<<<cuda_gridsize(size), BLOCK>>>(size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, s1, s2, out);
    check_error(hipPeekAtLastError());
}

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);
        if(abs_val < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2*abs_val - 1;
            delta[i] = (diff > 0) ? 1 : -1;
        }
    }
}

void smooth_l1_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    smooth_l1_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float t = truth[i];
        float p = pred[i];
        error[i] = (t) ? -log(p) : 0;
        delta[i] = t-p;
    }
}

void softmax_x_ent_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    softmax_x_ent_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float t = truth[i];
        float p = pred[i];
        error[i] = -t*log(p+.0000001) - (1-t)*log(1-p+.0000001);
        delta[i] = t-p;
    }
}

void logistic_x_ent_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    logistic_x_ent_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; //I know this is technically wrong, deal with it.
        delta[i] = diff;
    }
}

void l2_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    l2_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = abs(diff);
        delta[i] = (diff > 0) ? 1 : -1;
    }
}

void l1_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    l1_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}

__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        error[i] = truth[i] ? -pred[i] : pred[i];
        delta[i] = (truth[i] > 0) ? 1 : -1;
    }
}

void wgan_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    wgan_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}




__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        c[i] = s[i]*a[i] + (1-s[i])*(b ? b[i] : 0);
    }
}

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < (NX+NY)*B){
        int b = i / (NX+NY);
        int j = i % (NX+NY);
        if (j < NX){
            if(X) X[b*NX + j] += OUT[i];
        } else {
            if(Y) Y[b*NY + j - NX] += OUT[i];
        }
    }
}

void deinter_gpu(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    deinter_kernel<<<cuda_gridsize((NX+NY)*B), BLOCK>>>(NX, X, NY, Y, B, OUT);
    check_error(hipPeekAtLastError());
}

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < (NX+NY)*B){
        int b = i / (NX+NY);
        int j = i % (NX+NY);
        if (j < NX){
            OUT[i] = X[b*NX + j];
        } else {
            OUT[i] = Y[b*NY + j - NX];
        }
    }
}

void inter_gpu(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    inter_kernel<<<cuda_gridsize((NX+NY)*B), BLOCK>>>(NX, X, NY, Y, B, OUT);
    check_error(hipPeekAtLastError());
}

void weighted_sum_gpu(float *a, float *b, float *s, int num, float *c)
{
    weighted_sum_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, s, c);
    check_error(hipPeekAtLastError());
}

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        if(da) da[i] += dc[i] * s[i];
        if(db) db[i] += dc[i] * (1-s[i]);
        ds[i] += dc[i] * (a[i] - b[i]);
    }
}

void weighted_delta_gpu(float *a, float *b, float *s, float *da, float *db, float *ds, int num, float *dc)
{
    weighted_delta_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, s, da, db, ds, dc);
    check_error(hipPeekAtLastError());
}

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        c[i] += a[i]*b[i];
    }
}

void mult_add_into_gpu(int num, float *a, float *b, float *c)
{
    mult_add_into_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, c);
    check_error(hipPeekAtLastError());
}


__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        int val = input[i*stride];
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        float e = expf(input[i*stride]/temp - largest/temp);
        sum += e;
        output[i*stride] = e;
    }
    for(i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}


__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial*batch*groups) return;
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g]*spatial;
    int boff = b*stride;
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}

void softmax_tree(float *input, int spatial, int batch, int stride, float temp, float *output, tree hier)
{
    int *tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
    int *tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
    /*
       static int *tree_groups_size = 0;
       static int *tree_groups_offset = 0;
       if(!tree_groups_size){
       tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
       tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
       }
     */
    int num = spatial*batch*hier.groups;
    softmax_tree_kernel<<<cuda_gridsize(num), BLOCK>>>(input, spatial, batch, stride, temp, output, hier.groups, tree_groups_size, tree_groups_offset);
    check_error(hipPeekAtLastError());
    cuda_free((float *)tree_groups_size);
    cuda_free((float *)tree_groups_offset);
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch*groups) return;
    int b = id / groups;
    int g = id % groups;
    softmax_device(input + b*batch_offset + g*group_offset, n, temp, stride, output + b*batch_offset + g*group_offset);
}

void softmax_gpu(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    softmax_kernel<<<cuda_gridsize(batch*groups), BLOCK>>>(input, n, batch, batch_offset, groups, group_offset, stride, temp, output);
    check_error(hipPeekAtLastError());
}


__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int out_index = i;
    int out_w = i%(w*stride);
    i = i/(w*stride);
    int out_h = i%(h*stride);
    i = i/(h*stride);
    int out_c = i%c;
    i = i/c;
    int b = i%batch;

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b*w*h*c + in_c*w*h + in_h*w + in_w;


    if(forward) out[out_index] += scale * x[in_index];
    else atomicAdd(x+in_index, scale * out[out_index]);
}
void upsample_gpu(float *in, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t size = w*h*c*batch*stride*stride;
    upsample_kernel<<<cuda_gridsize(size), BLOCK>>>(size, in, w, h, c, batch, stride, forward, scale, out);
    check_error(hipPeekAtLastError());
}
